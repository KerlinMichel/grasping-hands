
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <set>
#include <limits>
#include <stdlib.h>
#include <ctime>
#include <chrono>
#include <thread>
#include <iomanip>
//#define INF std::numeric_limits<int>::max()
#define INF 2147483647
#define currTime duration_cast<milliseconds>(system_clock::now().time_since_epoch());

using namespace std::chrono;

template<std::size_t n>
__device__ void dijkstra(int *graph, int source, int target, int thread, bool* solved);

template<std::size_t n>
__device__ void Astar(int *graph, int source, int target, int thread, bool* solved);

template<std::size_t n>
void randomGraph(int (&graph)[n][n]);

template<std::size_t n>
void addCloseBias(int (&graph)[n][n]);

#define NUM_GRAPHS 1000

void stop(std::thread ts[NUM_GRAPHS])
{
  for(int i = 0; i < NUM_GRAPHS; ++i)
  {
    ts[i].join();
  }
}

int bg[NUM_GRAPHS][500][500];
int ss[NUM_GRAPHS];
int ts[NUM_GRAPHS];
__host__ __device__ inline int index(const int x, const int y, const int z) {
     return x * NUM_GRAPHS * NUM_GRAPHS + y * 500 + z;
}

__global__ void pathfind(int *g, int ts[NUM_GRAPHS], int ss[NUM_GRAPHS], bool *solved)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  idx++;
  printf("test\n");
  printf("%d", ss[0]);
  //pass in which graph is being worked on
  dijkstra<500>(g, ss[0], ts[0], 0, solved);
}

int main() 
{
  srand(time(NULL));
  bool* solved = (bool*) malloc(NUM_GRAPHS*sizeof(bool));
  for(int i = 0; i < NUM_GRAPHS; ++i)
  {
    solved[i] = false;
    randomGraph(bg[i]);
    addCloseBias(bg[i]);
    ss[i] = 0 + (rand() % (int)(499 - 0 + 1));
    ts[i] = 0 + (rand() % (int)(499 - 0 + 1));
  }
  bool* solved_gpu;
  hipMalloc(&solved_gpu, NUM_GRAPHS*sizeof(bool));
  hipMemcpy(solved_gpu, solved, NUM_GRAPHS*sizeof(bool), hipMemcpyHostToDevice);
  
  int *flatG = (int*) malloc(NUM_GRAPHS*500*500*sizeof(int));
  for(int x = 0; x < NUM_GRAPHS; ++x)
    for(int y = 0; y < 500; ++y)
      for(int z = 0; z < 500; ++z)
        flatG[x*500*500 + y*500 + z] = bg[x][y][z];
  
  int *g;
  hipMalloc(&g, NUM_GRAPHS*500*500*sizeof(int));
  hipMemcpy(g, flatG, NUM_GRAPHS*500*500*sizeof(int), hipMemcpyHostToDevice);
  
  std::cout << "source: " << ss[0] << std::endl;
  std::cout << "target: " << ts[0] << std::endl;
  
  int *ss_gpu;
  hipMalloc(&ss_gpu, NUM_GRAPHS*sizeof(int));
  hipMemcpy(ss_gpu, ss, NUM_GRAPHS*sizeof(int), hipMemcpyHostToDevice);
  
  int *ts_gpu;
  hipMalloc(&ts_gpu, NUM_GRAPHS*sizeof(int));
  hipMemcpy(ts_gpu, ts, NUM_GRAPHS*sizeof(int), hipMemcpyHostToDevice);
  pathfind<<<1,1>>>(g, ss_gpu, ts_gpu, solved_gpu);

  //sync to see print statements
  hipDeviceSynchronize();
  return 0;
}

__device__ bool setEmpty(bool* set, int size)
{
  for(int i = 0; i < size; ++i)
    if(set[i])
      return false;
  return true;
}

__device__ int firstSet(bool*set, int size)
{
  for(int i = 0; i < size; ++i)
    if(set[i])
      return i;
  return -1;
}

template<std::size_t n>
__device__ void dijkstra(int *graph, int source, int target, int thread, bool* solved)
{
  int *dist = new int[n];
  for(int i = 0; i < n; ++i)
    dist[i] = INF;
  dist[source] = 0;
  bool *vertices = new bool[n];
  for(int i = 0; i < n; i++)
  {
    vertices[i] = true;
  }
  while(!setEmpty(vertices, n))
  {
    if(solved[thread])
    {
      return;
    }
    int minV = firstSet(vertices, n);
    if(minV == -1)
      break;
    int min = dist[minV];
    for(int i = 0; i < n; ++i)
    {
      if(vertices[i] && dist[i] < min)
      {
        minV = i;
        min = dist[i];
      }
    }
    if(minV == target)
      break;
    vertices[minV] = false;
    for(int i = 0; i < n; ++i)
    {
      //printf("i:%d numV:%d\n", i, numV);
      //skip if not adjacent
      if(graph[index(0,minV,i)] == INF)
        continue;
      int newDist = dist[minV] + graph[index(0,minV,i)];//graph[minV][i];//dist.at(minV) + graph[minV][*it];
      if(newDist < dist[i])//dist.at(*it))
      {
        dist[i] = newDist;// dist.at(*it) = newDist;
      }
      //printf("end for loop\n");
    }
    //printf("end while loop, set empty:%d", setEmpty(vertices, n));
  }
  solved[thread] = true;
  //for(int i = 0; i < n; i++)
  //{
    printf("%d: | \n", dist[target]);
    //std::cout << i <<": " << dist.at(i) << " | "; 
  //}
}
#include <cmath>
template<std::size_t n>
__device__ void Astar(int *graph, int source, int target, int thread, bool* solved)
{
  int *dist = new int[n];
  for(int i = 0; i < n; ++i)
    dist[i] = INF;
  dist[source] = 0;
  bool *vertices = new bool[n];
  for(int i = 0; i < n; i++)
  {
    vertices[i] = true;
  }
  while(!setEmpty(vertices, n))
  {
    if(solved[thread])
    {
      return;
    }
    int minV = firstSet(vertices, n);
    if(minV == -1)
      break;
    int min = dist[minV];
    for(int i = 0; i < n; ++i)
    {
      if(vertices[i] && dist[i]+2*(std::abs(i-target)) < min)
      {
        minV = i;
        min = dist[i];
      }
    }
    if(minV == target)
      break;
    vertices[minV] = false;
    for(int i = 0; i < n; ++i)
    {
      //printf("i:%d numV:%d\n", i, numV);
      //skip if not adjacent
      if(graph[index(0,minV,i)] == INF)
        continue;
      int newDist = dist[minV] + graph[index(0,minV,i)];//graph[minV][i];//dist.at(minV) + graph[minV][*it];
      if(newDist < dist[i])//dist.at(*it))
      {
        dist[i] = newDist;// dist.at(*it) = newDist;
      }
      //printf("end for loop\n");
    }
    //printf("end while loop, set empty:%d", setEmpty(vertices, n));
  }
  solved[thread] = true;
}

template<std::size_t n>
void randomGraph(int (&graph)[n][n])
{
  int maxEdges = n*(n-1);
  int numEdges = n + (rand() % (int)(maxEdges - (n) + 1));
  for(int i = 0; i < n; ++i)
    for(int j = 0; j < n; ++j)
      graph[i][j] = INF;
  
  for(int i = 0; i < numEdges; ++i)
  {
    int v1 = (rand() % (int)(n));
    int v2 = (rand() % (int)(n));
    //don't allow loops
    if(v1 == v2)
    {
      i--;
      continue;
    }
    int weight = 1 + (rand() % (int)(1000 - 1 + 1));
    graph[v1][v2] = graph[v2][v1] = weight;
  }
}

template<std::size_t n>
void addCloseBias(int (&graph)[n][n])
{
  for(int v = 0; v < n-10; )
  {
    int w1 = 1 + (rand() % (int)(5 - 1 + 1));
    int w2 = 1 + (rand() % (int)(5 - 1 + 1));
    int w3 = 1 + (rand() % (int)(5 - 1 + 1));
    int v1 = (v+1) + (rand() % (int)((v + 10) - (v+1) + 1));
    int v2 = (v+2) + (rand() % (int)((v + 10) - (v+2) + 1));
    int v3 = (v+3) + (rand() % (int)((v + 10) - (v+3) + 1));
    graph[v][v1] = graph[v1][v] = w1;
    graph[v][v2] = graph[v2][v] = w2;
    graph[v][v3] = graph[v3][v] = w3;
    v += 1 + (rand() % (int)(5 - 1 + 1)); 
    if(v >= n)
      break;
  }  
}
